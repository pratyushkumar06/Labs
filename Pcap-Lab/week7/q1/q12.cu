#include "hip/hip_runtime.h"


#include <stdio.h>

#define M 6

__global__ void add (int *A, int *B, int *C) {
	int idx = threadIdx.x;
    printf("idx = %d\n", idx);
    C[idx] = A[idx] + B[idx];
}

int main () {
    int A[M], B[M], C[M];

    int i, j;
    for (i = 0; i < M; ++i) {
        A[i] = i + 1;
        B[i] = M - i - 1;
    }
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);
    add<<<1, M>>>(d_a, d_b, d_c);
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);
    printf("A:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", A[j]);
    }
    printf("\n");
    printf("B:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", B[j]);
    }
    printf("\n");
    printf("A + B:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", C[j]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	getchar();

    return 0;
}