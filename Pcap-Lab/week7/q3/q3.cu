#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void add (int *A, int *B,int *a) {
    int idx = blockIdx.x;
    printf("idx = %d\n", idx);
    A[idx] = (*a)*A[idx] + B[idx];
}

int main () {
    int M;
    int i, j;
    printf("Enter the size : ");
    scanf("%d",&M);
    int A[M], B[M];
    printf("Enter the X and Y values : \n");
    for (i = 0; i < M; ++i) { 
        scanf("%d%d",&A[i],&B[i]);
    }
    printf("Enter alpha : ");
    int a;
    scanf("%d",&a);
    int *d_a, *d_b, *d_c;
    int size = sizeof(int) * M;
    hipMalloc((void**)&d_c,sizeof(int));
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, &a, sizeof(int), hipMemcpyHostToDevice);
    add<<<M, 1>>>(d_a, d_b, d_c);
    hipMemcpy(&A, d_a, size, hipMemcpyDeviceToHost);
    printf("\n");
    printf("aA + B:\n");
    for (j = 0; j < M; ++j) {
        printf("%d\t", A[j]);
    }
    printf("\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}