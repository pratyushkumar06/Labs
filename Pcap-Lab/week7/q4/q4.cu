#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

__global__ void AddIntsCUDA(int *a, int *b,int *d) //Kernel Definition
{
 *a = (*a)*(*d) + *b;
}

int main()
{
 int a = 5, b = 9,d=2;
 int *d_a, *d_b,*d_d; //Device variable Declaration

        //Allocation of Device Variables 
 hipMalloc((void **)&d_a, sizeof(int));
 hipMalloc((void **)&d_b, sizeof(int));
 hipMalloc((void **)&d_d, sizeof(int));
        //Copy Host Memory to Device Memory
 hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(d_d, &d, sizeof(int), hipMemcpyHostToDevice);
        
        //Launch Kernel
 AddIntsCUDA << <1, 1 >> >(d_a, d_b,d_d);

        //Copy Device Memory to Host Memory
 hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

 printf("The answer is %d",a);

 
        //Free Device Memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_d);

 return 0;
}